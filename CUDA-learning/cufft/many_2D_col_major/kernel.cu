#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define NX 3335 // 有效数据个数
#define NXWITH0 5000
#define Nfft 128
#define BLOCK_SIZE 32
using std::cout;
using std::endl;


/**
* 功能：判断两个 cufftComplex 数组的是否相等
* 输入：idataA 输入数组A的头指针
* 输入：idataB 输出数组B的头指针
* 输入：size 数组的元素个数
* 返回：true | false
*/
bool IsEqual(hipfftComplex *idataA, hipfftComplex *idataB, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (abs(idataA[i].x - idataB[i].x) > 0.000001 || abs(idataA[i].y - idataB[i].y) > 0.000001)
			return false;
	}

	return true;
}


/**
* 功能：实现 cufftComplex 数组的尺度缩放，也就是乘以一个数
* 输入：idata 输入数组的头指针
* 输出：odata 输出数组的头指针
* 输入：size 数组的元素个数
* 输入：scale 缩放尺度
*/
__global__ void CufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, float scale)
{
	// 坐标索引
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // 列坐标
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // 行坐标
	int index = y_id * NXWITH0 + x_id;

	if (x_id < NX && y_id < Nfft)
	{
		odata[index].x = idata[index].x * scale;
		odata[index].y = idata[index].y * scale;
	}
}

int main()
{
	/* 开辟设备端的内存空间 */
	hipfftComplex *data_dev;
	hipMalloc((void**)&data_dev, Nfft * NXWITH0 * sizeof(hipfftComplex)); // 开辟设备内存
	hipMemset(data_dev, 0, Nfft * NXWITH0 * sizeof(hipfftComplex)); // 初始为0

	/* 开辟主机端的内存空间 */
	hipfftComplex *data_Host = (hipfftComplex*)malloc(Nfft * NXWITH0 * sizeof(hipfftComplex)); // 主机端数据头指针
	hipfftComplex *resultIFFT = (hipfftComplex*)malloc(Nfft * NXWITH0 * sizeof(hipfftComplex)); // 先正变换后逆变换的结果


	/* 初始数据 */
	for (int i = 0; i < Nfft; i++)
	{
		for (int j = 0; j < NXWITH0; j++)
		{
			int index = i * NXWITH0 + j;
			data_Host[index].x = float((rand() * rand()) % NX) / NX;
			data_Host[index].y = float((rand() * rand()) % NX) / NX;
		}
	}

	/* 线程块和线程格 */
	dim3 dimBlock2D(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid2D_NXWITH0_Nfft((NXWITH0 + BLOCK_SIZE - 1) / dimBlock2D.x, (Nfft + BLOCK_SIZE - 1) / dimBlock2D.y);

	/* 申请 cufft 句柄*/
	hipfftHandle plan_Nfft_Many; // 创建cuFFT句柄
	const int rank = 1; // 一维 fft
	int n[rank] = { Nfft }; // 进行 fft 的信号的长度为 Nfft
	int inembed[1] = { 0 }; // 输入数据的[页数，列数，行数]
	int onembed[1] = { 10 }; // 输出数据的[页数，列数，行数]
	int istride = NXWITH0; // 每个输入信号相邻两个元素的距离
	int idist = 1; // 每两个输入信号第一个元素的距离
	int ostride = NXWITH0; // 每个输出信号相邻两个元素的距离
	int odist = 1; // 每两个输出信号第一个元素的距离
	int batch = NX; // 进行 fft 的信号个数
	hipfftPlanMany(&plan_Nfft_Many, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	/* 核心部份 */

	hipMemcpy(data_dev, data_Host, Nfft * NXWITH0 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecC2C(plan_Nfft_Many, data_dev, data_dev, HIPFFT_FORWARD); // 执行 cuFFT，正变换
	hipfftExecC2C(plan_Nfft_Many, data_dev, data_dev, HIPFFT_BACKWARD); // 执行 cuFFT，逆变换
	CufftComplexScale<<<dimGrid2D_NXWITH0_Nfft, dimBlock2D>>>(data_dev, data_dev, 1.0f / Nfft); // 乘以系数
	hipMemcpy(resultIFFT, data_dev, Nfft * NXWITH0 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	/* 销毁句柄 */
	hipfftDestroy(plan_Nfft_Many);

	/* 释放设备空间 */
	hipFree(data_dev);


	/* 输出结果 */
	cout << IsEqual(data_Host, resultIFFT, NX * Nfft) << endl;

	return 0;
}