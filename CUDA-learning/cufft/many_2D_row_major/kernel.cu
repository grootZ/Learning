#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define NX 3335 // 有效数据个数
#define Nfft 128
#define BLOCK_SIZE 32
using std::cout;
using std::endl;


/**
* 功能：判断两个 cufftComplex 数组的是否相等
* 输入：idataA 输入数组A的头指针
* 输入：idataB 输出数组B的头指针
* 输入：size 数组的元素个数
* 返回：true | false
*/
bool IsEqual(hipfftComplex *idataA, hipfftComplex *idataB, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (abs(idataA[i].x - idataB[i].x) > 0.000001 || abs(idataA[i].y - idataB[i].y) > 0.000001)
			return false;
	}

	return true;
}



/**
* 功能：实现 cufftComplex 数组的尺度缩放，也就是乘以一个数
* 输入：idata 输入数组的头指针
* 输出：odata 输出数组的头指针
* 输入：size 数组的元素个数
* 输入：scale 缩放尺度
*/
__global__ void CufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, float scale)
{
	// 坐标索引
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // 列坐标
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // 行坐标
	int index = y_id * NX + x_id;

	if (x_id < NX && y_id < Nfft)
	{
		odata[index].x = idata[index].x * scale;
		odata[index].y = idata[index].y * scale;
	}
}

int main()
{
	/* 开辟设备端的内存空间 */
	hipfftComplex *data_dev;
	hipMalloc((void**)&data_dev, Nfft * NX * sizeof(hipfftComplex)); // 开辟设备内存
	hipMemset(data_dev, 0, Nfft * NX * sizeof(hipfftComplex)); // 初始为0

	/* 开辟主机端的内存空间 */
	hipfftComplex *data_Host = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // 主机端数据头指针
	hipfftComplex *resultFFT = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // 正变换的结果
	hipfftComplex *resultIFFT = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // 先正变换后逆变换的结果


	/* 初始数据 */
	for (int i = 0; i < Nfft * NX; i++)
	{
		data_Host[i].x = float((rand() * rand()) % NX) / NX;
		data_Host[i].y = float((rand() * rand()) % NX) / NX;
	}

	/* 线程块和线程格 */
	dim3 dimBlock2D(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid2D_NX_Nfft((NX + BLOCK_SIZE - 1) / dimBlock2D.x, (Nfft + BLOCK_SIZE - 1) / dimBlock2D.y);

	/* 申请 cufft 句柄*/
	hipfftHandle plan_NX_Many; // 创建cuFFT句柄
	int number_NX[1] = { NX };
	hipfftPlanMany(&plan_NX_Many, 1, number_NX, NULL, 1, NX, NULL, 1, NX, HIPFFT_C2C, Nfft);

	/* 核心部份 */

	hipMemcpy(data_dev, data_Host, Nfft * NX * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecC2C(plan_NX_Many, data_dev, data_dev, HIPFFT_FORWARD); // 执行 cuFFT，正变换
	hipfftExecC2C(plan_NX_Many, data_dev, data_dev, HIPFFT_BACKWARD); // 执行 cuFFT，逆变换
	CufftComplexScale<<<dimGrid2D_NX_Nfft, dimBlock2D>>>(data_dev, data_dev, 1.0f / NX); // 乘以系数
	hipMemcpy(resultIFFT, data_dev, Nfft * NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	/* 销毁句柄 */
	hipfftDestroy(plan_NX_Many);

	/* 释放设备空间 */
	hipFree(data_dev);


	/* 输出结果 */
	cout << IsEqual(data_Host, resultIFFT, NX * Nfft) << endl;

	return 0;
}